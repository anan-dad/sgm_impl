#include "hip/hip_runtime.h"
#include "CTensor.h"
#include "timer.h"
#include <cstring>
#include <cmath>



void unarycosts_pixelwise_euclidean(CTensor<float>& leftImg, CTensor<float>& rightImg, int x_size, int y_size, float* Disparity)
{
  for( int y = 0; y < y_size; ++y){
    for( int x = 0; x < x_size; ++x){

  	  float ed_temp = sqrtf((leftImg(x, y, 0) - rightImg(x, y, 0)) * (leftImg(x, y, 0) - rightImg(x, y, 0)) +
     	              	    (leftImg(x, y, 1) - rightImg(x, y, 1)) * (leftImg(x, y, 1) - rightImg(x, y, 1)) +
        	             	(leftImg(x, y, 2) - rightImg(x, y, 2)) * (leftImg(x, y, 2) - rightImg(x, y, 2)));

	  Disparity[y * x_size + x] = ed_temp;	
	}
  }
}




void unarycosts_L1(CTensor<float>& leftImg, CTensor<float>& rightImg, int x_size, int y_size, float* Disparity)
{


  for( int y = 0; y < y_size; ++y){
    for( int x = 0; x < x_size; ++x){
      int l1_init = 37485;  // max difference between two kernels
      int l1_temp = 0;

      for(int sC = 50; sC >= 0 ; sC--){  // max matching distance is 50(only from right side)
        for(int i = -3; i < 4; i++){
          for(int j = -3; j < 4; j++){
            if(0 <= x + i < x_size && 0 <= y + j < y_size && 0 <= x + i + sC < x_size)
            {  
			  l1_temp += fabs(leftImg(x+i, y+j, 0) - rightImg(x+sC+i, y+j, 0)) + 
              			 fabs(leftImg(x+i, y+j, 1) - rightImg(x+sC+i, y+j, 1)) + 
              			 fabs(leftImg(x+i, y+j, 2) - rightImg(x+sC+i, y+j, 2)); 



           }
          } 
        }
    
        if(l1_temp <= l1_init)
          l1_init = l1_temp;

        Disparity[(y * x_size + x) * 51 + sC] = l1_temp;
        l1_temp = 0;
      } 
    }
  }
}


void unarycosts_L2(CTensor<float>& leftImg, CTensor<float>& rightImg, int x_size, int y_size, float* Disparity)
{

  for( int y = 0; y < y_size; ++y){
    for( int x = 0; x < x_size; ++x){

      int l2_init = 255 * 255 * 3 * 49 ;  // max difference between two kernels
      int l2_temp = 0;

      for(int sC = 0; sC <51; sC++){  // max matching distance is 50(only from right side)
        for(int i = -3; i < 4; i++){
          for(int j = -3; j < 4; j++){
            if(0 <= x + i < x_size && 0 <= y + j < y_size && 0 <= x + i + sC < x_size)
            {  
			  l2_temp += (leftImg(x+i ,y+j, 0) - rightImg(x+sC+i, y+j, 0)) * (leftImg(x+i ,y+j, 0) - rightImg(x+sC+i, y+j, 0)) +
						 (leftImg(x+i ,y+j, 1) - rightImg(x+sC+i, y+j, 1)) * (leftImg(x+i ,y+j, 1) - rightImg(x+sC+i, y+j, 1)) +
						 (leftImg(x+i ,y+j, 2) - rightImg(x+sC+i, y+j, 2)) * (leftImg(x+i ,y+j, 2) - rightImg(x+sC+i, y+j, 2));
            }
          } 
        }
    
        if(l2_temp <= l2_init)
          l2_init = l2_temp;

        Disparity[(y * x_size + x) * 51 + sC] = l2_temp;
        l2_temp = 0;
      } 
    }
  }

}

void unarycosts_NCC(CTensor<float>& leftImg, CTensor<float>& rightImg, int x_size, int y_size, float* Disparity)
{

  for( int y = 0; y < y_size; ++y){
    for( int x = 0; x < x_size; ++x){
      float ncc_init = 0;  // min difference bewteen two kernels

      for(int sC = 50; sC >= 0; sC--){  // max matching distance is 50(only from right side)
        int count = 0;
        float sum_left_x = 0;
        float sum_left_y = 0;
        float sum_left_z = 0;

        float sum_right_x = 0;
        float sum_right_y = 0;
        float sum_right_z = 0;
 
        for(int i = -3; i < 4; i++)
          for(int j = -3; j < 4; j++)
            if(0 <= x + i < x_size && 0 <= y + j < y_size){
              count ++;     
              sum_left_x += leftImg(x+i,y+j,0);
              sum_left_y += leftImg(x+i,y+j,1);
              sum_left_z += leftImg(x+i,y+j,2); 

              sum_right_x += rightImg(x+sC+i, y+j, 0);
              sum_right_y += rightImg(x+sC+i, y+j, 1);
              sum_right_z += rightImg(x+sC+i, y+j, 2);       
           }
  
        float average_left_x = sum_left_x / count;
        float average_left_y = sum_left_y / count;
        float average_left_z = sum_left_z / count;
  
        float average_right_x = sum_right_x / count;
        float average_right_y = sum_right_y / count;
        float average_right_z = sum_right_z / count;

        float ncc_temp1 = 0;
        float ncc_temp2 = 0;
        float ncc_temp  = 0;

        for(int i = -3; i < 4; i++){
          for(int j = -3; j < 4; j++){
            if(0 <= x + i < x_size && 0 <= y + j < y_size){
              ncc_temp1 += ((leftImg(x+i,y+j,0) - average_left_x)*(rightImg(x+sC+i, y+j, 0) - average_right_x) +
                            (leftImg(x+i,y+j,1) - average_left_y)*(rightImg(x+sC+i, y+j, 1) - average_right_y) +
                            (leftImg(x+i,y+j,2) - average_left_z)*(rightImg(x+sC+i, y+j, 2) - average_right_z));
  
              ncc_temp2 += sqrt(((leftImg(x+i,y+j,0) - average_left_x) * (leftImg(x+i,y+j,0) - average_left_x) +
                                 (leftImg(x+i,y+j,1) - average_left_y) * (leftImg(x+i,y+j,1) - average_left_y) +
                                 (leftImg(x+i,y+j,2) - average_left_z) * (leftImg(x+i,y+j,2) - average_left_z)) *
                                ((rightImg(x+sC+i, y+j, 0) - average_right_x) * (rightImg(x+sC+i, y+j, 0) - average_right_x) +
                                 (rightImg(x+sC+i, y+j, 1) - average_right_y) * (rightImg(x+sC+i, y+j, 1) - average_right_y) +
                                 (rightImg(x+sC+i, y+j, 2) - average_right_z) * (rightImg(x+sC+i, y+j, 2) - average_right_z)));
              
              ncc_temp = ncc_temp1 / ncc_temp2;  
            }
          }
        }

        if(ncc_temp >= ncc_init)
            ncc_init = ncc_temp;

        Disparity[(y * x_size + x) * 51 + sC] = 10000*(1 - ncc_temp);
        ncc_temp = 0;
      }
    }
  }
}




void belief_propagation(int x_size, int y_size, float* MpqsF0, float* MpqsB0, float* MpqsF2, float* MpqsB2, float* MpqsF4, float* MpqsB4, float* MpqsF6, float* MpqsB6, float* MpqsF9, float* MpqsB9,
                                                           float* MpqsF11, float* MpqsB11, float* MpqsF13, float* MpqsB13, float* MpqsF15, float* MpqsB15, float* Disparity, float* result)
{

  int potts = 0;
  int lambda0 = 1500, lambda2 = 1000, lambda4 = 1000, lambda6 = 1000, lambda9 = 1000, lambda11 = 1000, lambda13 = 1000, lambda15 = 1000;





  //forward pass 0 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++)
      MpqsF0[y * x_size * 51 + j] = 0.0f;
    for(int q = 1; q < x_size; q++)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        MpqsF0[(y * x_size + q) * 51 + j] = Disparity[(y * x_size + q-1) * 51] + MpqsF0[(y * x_size + q-1) * 51] + lambda0 * potts;
        for(int i = 1; i <= 50; i++)
        {
          if(i == j)
            potts = 0;
          else
            potts = 1;
          float costf0 = Disparity[(y * x_size + q-1) * 51 + i] + MpqsF0[(y * x_size + q-1) * 51 + i] + lambda0 * potts;
          if(costf0 < MpqsF0[(y * x_size + q) * 51 + j])
            MpqsF0[(y * x_size + q) * 51 + j] = costf0;
        }
      }
    }  
    
  //backward pass 0 degree
    for(int j = 0; j <= 50; j++)
      MpqsB0[((y+1) * x_size - 1) * 51 + j] = 0.0f;
    for(int q = x_size - 2; q >= 0; q--)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        MpqsB0[(y * x_size + q) * 51 + j] = Disparity[(y * x_size + q+1) * 51] + MpqsB0[(y * x_size + q+1) * 51] + lambda0 * potts;
        for(int i = 1; i <= 50; i++)
        {
          if(i == j)
            potts = 0;
          else
            potts = 1;
          float costb0 = Disparity[(y * x_size + q+1) * 51 + i] + MpqsB0[(y * x_size + q+1) * 51 + i] + lambda0 * potts;
          if(costb0 < MpqsB0[(y * x_size + q) * 51 + j])
            MpqsB0[(y * x_size + q) * 51 + j] = costb0;
        }
      }
    }
  }



  //forward pass 22.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsF2[y * x_size * 51 + j] = 0.0f;
      MpqsF2[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF2[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF2[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }
  

  for( int x = 0; x < x_size; ++x){  
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+1) * x_size + x-2 < 0 || (q+1) * x_size + x-2 >= y_size * x_size) 
          MpqsF2[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF2[(q * x_size + x) * 51 + j] = Disparity[((q+1) * x_size + x-2) * 51] + MpqsF2[((q+1) * x_size + x-2) * 51] + lambda2 * potts;       
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf2 = Disparity[((q+1) * x_size + x-2) * 51 + i] + MpqsF2[((q+1) * x_size + x-2) * 51 + i] + lambda2 * potts;
            if(costf2 < MpqsF2[(q * x_size + x) * 51 + j])
              MpqsF2[(q * x_size + x) * 51 + j] = costf2;
          }
	    }
      }
    }  
  }


  //backward pass 22.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsB2[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsB2[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB2[j] = 0.0f;
  
  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-1) * x_size + x+2 < 0 || (q-1) * x_size + x+2 >= y_size * x_size)
          MpqsB2[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB2[(q * x_size + x) * 51 + j] = Disparity[((q-1) * x_size + x+2) * 51] + MpqsB2[((q-1) * x_size + x+2) * 51] + lambda2 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb2 = Disparity[((q-1) * x_size + x+2) * 51 + i] + MpqsB2[((q-1) * x_size + x+2) * 51 + i] + lambda2 * potts;
            if(costb2 < MpqsB2[(q * x_size + x) * 51 + j])
              MpqsB2[(q * x_size + x) * 51 + j] = costb2;
          }
        }
      }
    }
  }  


  

  

  //forward pass 45 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsF4[y * x_size * 51 + j] = 0.0f;
      MpqsF4[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF4[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF4[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }

  for( int x = 0; x < x_size; ++x){
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+1) * x_size + x-1 < 0 || (q+1) * x_size + x-1 >= y_size * x_size) 
          MpqsF4[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF4[(q * x_size + x) * 51 + j] = Disparity[((q+1) * x_size + x-1) * 51] + MpqsF4[((q+1) * x_size + x-1) * 51] + lambda4 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf4 = Disparity[((q+1) * x_size + x-1) * 51 + i] + MpqsF4[((q+1) * x_size + x-1) * 51 + i] + lambda4 * potts;
            if(costf4 < MpqsF4[(q * x_size + x) * 51 + j])
              MpqsF4[(q * x_size + x) * 51 + j] = costf4;
          }
        }
      }  
    }  
  }


  //backward pass 45 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsB4[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsB4[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB4[j] = 0.0f;

  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-1) * x_size + x+1 < 0 || (q-1) * x_size + x+1 >= y_size * x_size)
          MpqsB4[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB4[(q * x_size + x) * 51 + j] = Disparity[((q-1) * x_size + x+1) * 51] + MpqsB4[((q-1) * x_size + x+1) * 51] + lambda4 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb4 = Disparity[((q-1) * x_size + x+1) * 51 + i] + MpqsB4[((q-1) * x_size + x+1) * 51 + i] + lambda4 * potts;
            if(costb4 < MpqsB4[(q * x_size + x) * 51 + j])
              MpqsB4[(q * x_size + x) * 51 + j] = costb4;
          }
        }
      }
    }
  }




  //forward pass 67.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsF6[y * x_size * 51 + j] = 0.0f;
      MpqsF6[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF6[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF6[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }

  for( int x = 0; x < x_size; ++x){ 
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+2) * x_size + x-1 < 0 || (q+2) * x_size + x-1 >= y_size * x_size) 
          MpqsF6[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF6[(q * x_size + x) * 51 + j] = Disparity[((q+2) * x_size + x-1) * 51] + MpqsF6[((q+2) * x_size + x-1) * 51] + lambda6 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf6 = Disparity[((q+2) * x_size + x-1) * 51 + i] + MpqsF6[((q+2) * x_size + x-1) * 51 + i] + lambda6 * potts;
            if(costf6 < MpqsF6[(q * x_size + x) * 51 + j])
              MpqsF6[(q * x_size + x) * 51 + j] = costf6;
          }
        }
      }   
    }   
  }

  //backward pass 67.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsB6[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsB6[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB6[j] = 0.0f;
 
  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-2) * x_size + x+1 < 0 || (q-2) * x_size + x+1 >= y_size * x_size)
          MpqsB6[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB6[(q * x_size + x) * 51 + j] = Disparity[((q-2) * x_size + x+1) * 51] + MpqsB6[((q-2) * x_size + x+1) * 51] + lambda6 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb6 = Disparity[((q-2) * x_size + x+1) * 51 + i] + MpqsB6[((q-2) * x_size + x+1) * 51 + i] + lambda6 * potts;
            if(costb6 < MpqsB6[(q * x_size + x) * 51 + j])
              MpqsB6[(q * x_size + x) * 51 + j] = costb6;
          }
        }
      }
    }
  }






  //forward pass 90 degree
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF9[((y_size - 2) * x_size) * 51 + j] = 0.0f;
  }

  for( int x = 0; x < x_size; ++x){
    for(int q = y_size - 2; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        MpqsF9[(q * x_size + x) * 51 + j] = Disparity[((q+1) * x_size + x) * 51] + MpqsF9[((q+1) * x_size + x) * 51] + lambda9 * potts;
        for(int i = 1; i <= 50; i++)
        {
          if(i == j)
            potts = 0;
          else
            potts = 1;
          float costf9 = Disparity[((q+1) * x_size + x) * 51 + i] + MpqsF9[((q+1) * x_size + x) * 51 + i] + lambda9 * potts;
          if(costf9 < MpqsF9[((q+1) * x_size + x) * 51 + j])
            MpqsF9[((q+1) * x_size + x) * 51 + j] = costf9;
        }
      }
    }  
  }

  //backward pass 90 degree
  for(int j = 0; j <= (x_size-1)*51; j++)
    MpqsB9[j] = 0.0f;

  for( int x = 0; x < x_size; ++x){
    for(int q = 1; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        MpqsB9[(q * x_size + x) * 51 + j] = Disparity[((q-1) * x_size + x) * 51] + MpqsB9[((q-1) * x_size + x) * 51] + lambda9 * potts;
        for(int i = 1; i <= 50; i++)
        {
          if(i == j)
            potts = 0;
          else
            potts = 1;
          float costb9 = Disparity[((q-1) * x_size + x) * 51 + i] + MpqsB9[((q-1) * x_size + x) * 51 + i] + lambda9 *potts;
          if(costb9 < MpqsB9[((q-1) * x_size + x) * 51 + j])
            MpqsB9[((q-1) * x_size + x) * 51 + j] = costb9;
        }
      }
    }
  }




  //forward pass 112.5 degree


  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsF11[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsF11[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF11[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF11[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }


  for( int x = 0; x < x_size; ++x){
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+2) * x_size + x+1 < 0 || (q+2) * x_size + x+1 >= y_size * x_size) 
          MpqsF11[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF11[(q * x_size + x) * 51 + j] = Disparity[((q+2) * x_size + x+1) * 51] + MpqsF11[((q+2) * x_size + x+1) * 51] + lambda11 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf11 = Disparity[((q+2) * x_size + x+1) * 51 + i] + MpqsF11[((q+2) * x_size + x+1) * 51 + i] + lambda11 * potts;
            if(costf11 < MpqsF11[(q * x_size + x) * 51 + j])
              MpqsF11[(q * x_size + x) * 51 + j] = costf11;
          }
        }
      }   
    }  
  }

  //backward pass 112.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsB11[y * x_size * 51 + j] = 0.0f;
      MpqsB11[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB11[j] = 0.0f;

  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-2) * x_size + x-1 < 0 || (q-2) * x_size + x-1 >= y_size * x_size)
          MpqsB11[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB11[(q * x_size + x) * 51 + j] = Disparity[((q-2) * x_size + x-1) * 51] + MpqsB11[((q-2) * x_size + x-1) * 51] + lambda11 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb11 = Disparity[((q-2) * x_size + x-1) * 51 + i] + MpqsB11[((q-2) * x_size + x-1) * 51 + i] + lambda11 * potts;
            if(costb11 < MpqsB11[(q * x_size + x) * 51 + j])
              MpqsB11[(q * x_size + x) * 51 + j] = costb11;
          }
        }
      }
    }
  }


  //forward pass 135 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsF13[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsF13[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF13[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF13[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }

  for( int x = 0; x < x_size; ++x){
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+1) * x_size + x+1 < 0 || (q+1) * x_size + x+1 >= y_size * x_size) 
          MpqsF13[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF13[(q * x_size + x) * 51 + j] = Disparity[((q+1) * x_size + x+1) * 51] + MpqsF13[((q+1) * x_size + x+1) * 51] + lambda13 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf13 = Disparity[((q+1) * x_size + x+1) * 51 + i] + MpqsF13[((q+1) * x_size + x+1) * 51 + i] + lambda13 * potts;
            if(costf13 < MpqsF13[(q * x_size + x) * 51 + j]) 
              MpqsF13[(q * x_size + x) * 51 + j] = costf13;
          }
        }
      }  
    }  
  }

              

  //backward pass 135 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsB13[y * x_size * 51 + j] = 0.0f;
      MpqsB13[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB13[j] = 0.0f;

  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-1) * x_size + x-1 < 0 || (q-1) * x_size + x-1 >= y_size * x_size)
          MpqsB13[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB13[(q * x_size + x) * 51 + j] = Disparity[((q-1) * x_size + x-1) * 51] + MpqsB13[((q-1) * x_size + x-1) * 51] + lambda13 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb13 = Disparity[((q-1) * x_size + x-1) * 51 + i] + MpqsB13[((q-1) * x_size + x-1) * 51 + i] + lambda13 * potts;
            if(costb13 < MpqsB13[(q * x_size + x) * 51 + j])
              MpqsB13[(q * x_size + x) * 51 + j] = costb13;
          }
        }
      }
    }
  }



 


  //forward pass 157.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50; j++){
      MpqsF15[((y+1) * x_size - 1) * 51 + j] = 0.0f;
      MpqsF15[((y+1) * x_size - 2) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= (x_size-1) * 51 ; j++){
    MpqsF15[((y_size - 2) * x_size) * 51 + j] = 0.0f;
    MpqsF15[((y_size - 3) * x_size) * 51 + j] = 0.0f;
  }
 
  for( int x = 0; x < x_size; ++x){
    for(int q = y_size - 3; q >= 0; q--)
    {
      for(int j = 0; j <= 50 ; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q+1) * x_size + x+2 < 0 || (q+1) * x_size + x+2 >= y_size * x_size) 
          MpqsF15[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsF15[(q * x_size + x) * 51 + j] = Disparity[((q+1) * x_size + x+2) * 51] + MpqsF15[((q+1) * x_size + x+2) * 51] + lambda15 * potts;
          for(int i = 1; i <= 50; i++)
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costf15 = Disparity[((q+1) * x_size + x+2) * 51 + i] + MpqsF15[((q+1) * x_size + x+2) * 51 + i] + lambda15 * potts;
            if(costf15 < MpqsF15[(q * x_size + x) * 51 + j])
              MpqsF15[(q * x_size + x) * 51 + j] = costf15;
          }
        }
      }  
    }  
  }


  //backward pass 157.5 degree
  for( int y = 0; y < y_size; ++y){
    for(int j = 0; j <= 50 ; j++){
      MpqsB15[y * x_size * 51 + j] = 0.0f;
      MpqsB15[(y * x_size + 1) * 51 + j] = 0.0f;
    }
  }
  for(int j = 0; j <= 2*(x_size-1)*51; j++)
    MpqsB15[j] = 0.0f;

  for( int x = 0; x < x_size; ++x){
    for(int q = 2; q < y_size; q++)
    {
      for(int j = 0; j <= 50; j++)
      {
        if(j == 0)
          potts = 0;
        else
          potts = 1;
        if((q-1) * x_size + x-2 < 0 || (q-1) * x_size + x-2 >= y_size * x_size)
          MpqsB15[(q * x_size + x) * 51 + j] = 0;
        else{
          MpqsB15[(q * x_size + x) * 51 + j] = Disparity[((q-1) * x_size + x-2) * 51] + MpqsB15[((q-1) * x_size + x-2) * 51] + lambda15 * potts;
          for(int i = 1; i <= 50; i++)
    
          {
            if(i == j)
              potts = 0;
            else
              potts = 1;
            float costb15 = Disparity[((q-1) * x_size + x-2) * 51 + i] + MpqsB15[((q-1) * x_size + x-2) * 51 + i] + lambda15 * potts;
            if(costb15 < MpqsB15[(q * x_size + x) * 51 + j])
              MpqsB15[(q * x_size + x) * 51 + j] = costb15;
          }
        }
      }
    }
  }



  //decision
  int minIndex = 0;

  for( int y = 0; y < y_size; ++y){
    for(int q = 0; q < x_size; q++)
    {
      minIndex = 0;

      float minCost = Disparity[(y * x_size + q) * 51] + MpqsF0[(y * x_size + q) * 51] + MpqsB0[(y * x_size + q) * 51] + MpqsF2[(y * x_size + q) * 51] + MpqsB2[(y * x_size + q) * 51]
                                                       + MpqsF4[(y * x_size + q) * 51] + MpqsB4[(y * x_size + q) * 51] + MpqsF6[(y * x_size + q) * 51] + MpqsB6[(y * x_size + q) * 51]
                                                       + MpqsF9[(y * x_size + q) * 51] + MpqsB9[(y * x_size + q) * 51] + MpqsF11[(y * x_size + q) * 51] + MpqsB11[(y * x_size + q) * 51]
                                                       + MpqsF13[(y * x_size + q) * 51] + MpqsB13[(y * x_size + q) * 51] + MpqsF15[(y * x_size + q) * 51] + MpqsB15[(y * x_size + q) * 51];

      for(int i = 1; i <= 50; i++)
      { 
        float cost = Disparity[(y * x_size + q) * 51 + i] + MpqsF0[(y * x_size + q) * 51 + i] + MpqsB0[(y * x_size + q) * 51 + i] + MpqsF2[(y * x_size + q) * 51 + i] + MpqsB2[(y * x_size + q) * 51 + i]
                                                          + MpqsF4[(y * x_size + q) * 51 + i] + MpqsB4[(y * x_size + q) * 51 + i] + MpqsF6[(y * x_size + q) * 51 + i] + MpqsB6[(y * x_size + q) * 51 + i]
                                                          + MpqsF9[(y * x_size + q) * 51 + i] + MpqsB9[(y * x_size + q) * 51 + i] + MpqsF11[(y * x_size + q) * 51 + i] + MpqsB11[(y * x_size + q) * 51 + i]
                                                          + MpqsF13[(y * x_size + q) * 51 + i] + MpqsB13[(y * x_size + q) * 51 + i] + MpqsF15[(y * x_size + q) * 51 + i] + MpqsB15[(y * x_size + q) * 51 + i];
        if(cost < minCost)
        {
          minCost = cost;
          minIndex = i;
        }
      }

      result[y * x_size + q] = minIndex;

    }
  } 
} 


void imgconv(int x_size, int y_size, float* result, CTensor<float>& resultImg)
{
  for( int y = 0; y < y_size; ++y)
    for( int x = 0; x < x_size; ++x){

  int sC = result[y * x_size + x];
  float a = sC * 5;
  if(a > 255 || a < 0)
    a = fmax(0, fmin(a, 255));

  resultImg(x,y,0) = a;
  resultImg(x,y,1) = a;
  resultImg(x,y,2) = a;

}

}


int main(int argc, char** argv)
{
  
  /*-----------------------------------------------------------------------
   *  Read rectified left and right input image and put them into
   *  Color CMatrices
   *-----------------------------------------------------------------------*/
  CTensor<float> leftImg;
  leftImg.readFromPPM("couchR.ppm");

  CTensor<float> rightImg;
  rightImg.readFromPPM("couchL.ppm");

  CTensor<float> resultImg;
  resultImg.readFromPPM("couchL.ppm");


  float* Disparity = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* imgStore0 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize());
  float* imgStore1 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize());
  float* imgStore2 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize());

  float* MpqsF0 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB0 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF2 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB2 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF4 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB4 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF6 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB6 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF9 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB9 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF11 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB11 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF13 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB13 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsF15 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* MpqsB15 = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize()*51);

  float* result = (float*) malloc(sizeof(float)*leftImg.xSize()*leftImg.ySize());

  // Which block matching method do we choose?
  int input;
  do {
    std::cout << "\n"
              << "Choose:\n\n"
              << " (1): Sum of absolute differences(L1)\n"
              << " (2): Sum of squared differences(L2)\n"
              << " (3): Normalized Cross Correlation\n"
              << "\n"
              << "Our choice [1-3]: ";
    std::cin >> input;
  } while (input < 1 or input > 3);

  switch (input) {
   // 
	case 1: { unarycosts_L1(leftImg, rightImg, leftImg.xSize(), leftImg.ySize(), Disparity);   break; }    
	case 2: { unarycosts_L2(leftImg, rightImg, leftImg.xSize(), leftImg.ySize(), Disparity);   break; }
    case 3: { unarycosts_NCC(leftImg, rightImg, leftImg.xSize(), leftImg.ySize(), Disparity);   break; }
    default: throw std::runtime_error("Invalid choice");
  }
  
 
  timer::start("CPU processing");
  belief_propagation(leftImg.xSize(), leftImg.ySize(), MpqsF0, MpqsB0, MpqsF2, MpqsB2, MpqsF4, MpqsB4, MpqsF6, MpqsB6, MpqsF9, MpqsB9, 
                                                       MpqsF11, MpqsB11, MpqsF13, MpqsB13, MpqsF15, MpqsB15, Disparity, result);
  
  imgconv(leftImg.xSize(), leftImg.ySize(), result, resultImg);

  resultImg.writeToPPM("couch.ppm");
  timer::stop("CPU processing"); 
  timer::printToScreen();



  free(Disparity);
  free(MpqsF0);
  free(MpqsB0);
  free(MpqsF2);
  free(MpqsB2);
  free(MpqsF4);
  free(MpqsB4);
  free(MpqsF6);
  free(MpqsB6);
  free(MpqsF9);
  free(MpqsB9);
  free(MpqsF11);
  free(MpqsB11);
  free(MpqsF13);
  free(MpqsB13);
  free(MpqsF15);
  free(MpqsB15);
  free(result);

  return 0;
}
